#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define pixel(i, j, w)  (((j)*(w)) +(i))
#define TPB 128 // THREADS PER BLOCK needs testing

int max_num_triangles;

/* A point in 2D space */
struct Point {
    double x;
    double y;
    double value;
};

/* A triangle defined by three points */
struct Triangle {
    struct Point p1;
    struct Point p2;
    struct Point p3;
};

/* Helper function to output the triangles in the Delaunay Triangulation */
void print_triangles(struct Triangle * triangles, int num_triangles) {
    for (int i = 0; i < num_triangles; i++) {
        printf("(%lf, %lf) (%lf, %lf) (%lf, %lf)\n", 
            triangles[i].p1.x, triangles[i].p1.y,
            triangles[i].p2.x, triangles[i].p2.y,
            triangles[i].p3.x, triangles[i].p3.y);   
    }
}

/* Helper function to calculate the distance between two points */
__device__ double distance(struct Point * p1, struct Point * p2) {
    double dx = (*p1).x - (*p2).x;
    double dy = (*p1).y - (*p2).y;
    return sqrt(dx*dx + dy*dy);
}

/* Helper function to check if a triangle is clockwise */
__device__ int is_ccw(struct Triangle * t) {
    double ax = (*t).p2.x - (*t).p1.x;
    double ay = (*t).p2.y - (*t).p1.y;
    double bx = (*t).p3.x - (*t).p1.x;
    double by = (*t).p3.y - (*t).p1.y;

    double area = ax * by - ay * bx;
    return area > 0;
}

/* Helper function to check if a point is inside a circle defined by three points */
__device__ int inside_circle(struct Point * p, struct Triangle * t) {
//      | ax-dx, ay-dy, (ax-dx)² + (ay-dy)² |
//det = | bx-dx, by-dy, (bx-dx)² + (by-dy)² |
//      | cx-dx, cy-dy, (cx-dx)² + (cy-dy)² |

    int clockwise = is_ccw(t);
    
    double ax = (*t).p1.x - (*p).x;
    double ay = (*t).p1.y - (*p).y;
    double bx = (*t).p2.x - (*p).x;
    double by = (*t).p2.y - (*p).y;
    double cx = (*t).p3.x - (*p).x;
    double cy = (*t).p3.y - (*p).y;

    double det = ax*by + bx*cy + cx*ay - ay*bx - by*cx - cy*ax;
    det = (ax*ax + ay*ay) * (bx*cy-cx*by) -
            (bx*bx + by*by) * (ax*cy-cx*ay) +
            (cx*cx + cy*cy) * (ax*by-bx*ay);
    
    if(clockwise) {
        return det > 0;
	}
    return det < 0;
}

//* Helper function to compute barycentric coordintaes of a point respect a triangle */
__device__ void barycentric_coordinates(struct Triangle * t, struct Point * p, double * alpha, double * beta, double * gamma) {
    // Compute the barycentric coordinates of the point with respect to the triangle
    (*alpha) = (((*t).p2.y - (*t).p3.y) * ((*p).x - (*t).p3.x) + ((*t).p3.x - (*t).p2.x) * ((*p).y - (*t).p3.y)) /
                  (((*t).p2.y - (*t).p3.y) * ((*t).p1.x - (*t).p3.x) + ((*t).p3.x - (*t).p2.x) * ((*t).p1.y - (*t).p3.y));
    (*beta) = (((*t).p3.y - (*t).p1.y) * ((*p).x - (*t).p3.x) + ((*t).p1.x - (*t).p3.x) * ((*p).y - (*t).p3.y)) /
                 (((*t).p2.y - (*t).p3.y) * ((*t).p1.x - (*t).p3.x) + ((*t).p3.x - (*t).p2.x) * ((*t).p1.y - (*t).p3.y));
    (*alpha) =(*alpha) > 0 ? (*alpha) : 0;
    (*alpha) =(*alpha) < 1 ? (*alpha) : 1;
    (*beta) = (*beta) > 0 ? (*beta) : 0;
    (*beta) = (*beta) < 1 ? (*beta) : 1;
    (*gamma) = 1.0 - (*alpha) - (*beta);
    (*gamma) = (*gamma) > 0 ? (*gamma) : 0;
    (*gamma) = (*gamma) < 1 ? (*gamma) : 1;
}


/* Helper function to check if a point is inside a triangle (IT CAN BE REMOVED)*/
__device__ int inside_triangle(struct Triangle * t, struct Point * p) {
    double alpha, beta, gamma;
    barycentric_coordinates(t, p, &alpha, &beta, &gamma); 
    // Check if the barycentric coordinates are positive and add up to 1
    if (alpha > 0 && beta > 0 && gamma > 0) {
        return 1;
    } else {
        return 0;
    }
}

/* Checks if p2 is in a square of size 5 around p1*/
__device__ int inside_square(struct Point *p1, struct Point *p2) {
	return (abs((p1->x - p2->x)) <= 2.5 && abs((p1->y - p2->y)) <= 2.5);
}

/* Helper function to save an image */   
void save_image(char const * filename, int width, int height, double *image){

   FILE *fp=NULL;
   fp = fopen(filename,"w");
   for(int j=0; j<height; ++j){
      for(int i=0; i<width; ++i){
         fprintf(fp,"%f ", image[pixel(i,j,width)]);      
      }
      fprintf(fp,"\n");
   }
   fclose(fp);

}

/* helper function to initialize the points */
void init_points(struct Point* points, int num_points, int width, int height) {
    for(int i = 0; i < num_points; i++) {
        points[i].x =  ((double) rand() / RAND_MAX)*width;
        points[i].y =  ((double) rand() / RAND_MAX)*height;
        points[i].value = 0.f;//(rand() % 10000) / 100.;
        //printf("Point %d [%f,%f]=%f\n", i, points[i].x, points[i].y, points[i].value);
    }
}

__global__ void count_close_points(struct Point* points, int num_points) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < num_points) {
		for(int i=0; i<idx; i++) {
			if (distance(&points[idx], &points[i]) < 100.f) {
				points[idx].value++;
			}
		}
		for(int i=idx+1; i<num_points; i++) {
			if (distance(&points[idx], &points[i]) < 100.f) {
				points[idx].value++;
			}
		}
	}
}

/*Wraper function to launch the CUDA kernel to count the close points*/
void count_close_points_gpu(struct Point* points, int num_points) {
	struct Point* d_points;
	size_t size = num_points * sizeof(struct Point);

	hipMalloc((void**) &d_points, size);

	hipMemcpy(d_points, points, size, hipMemcpyHostToDevice);

	int dimGrid = (num_points + (TPB-1)) / TPB; // amount of blocks of size TPB
	int dimBlock = TPB; // int multiple of 32 (warp size) (1024 maximum) try values 128-512

	count_close_points<<<dimGrid, dimBlock>>>(d_points, num_points);

	hipDeviceSynchronize();

	hipMemcpy(points, d_points, size, hipMemcpyDeviceToHost);
	hipFree(d_points);
}

__global__ void delaunay_triangulation(struct Point* points, int num_points, struct Triangle* triangles, int* num_triangles) {
	/* Iterate over every possible triangle defined by three points */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int i_max = num_points;
	int j_max = num_points-1;
	int k_max = num_points-2;

	int i = (idx/(j_max*k_max));
	int j = (idx/k_max) % (j_max) + 1;
	int k = idx % (k_max) + 2;

	if (i<i_max && j<j_max+1 && k<k_max+2 && i<j && j<k) {
		int flag = 0;
		struct Triangle local;
		local.p1 = points[i];
		local.p2 = points[j];
		local.p3 = points[k];
		
		for(int l=0; l<num_points; l++)	{
			if(inside_circle(&points[l], &local)) {
				flag = 1;
				break;
			}
		}

		if(flag == 0) {
			atomicAdd(num_triangles, 1);
			triangles[*num_triangles] = local;
		}
	}
}

/*Wraper function to launch the CUDA kernel to compute delaunay triangulation*/
void delaunay_triangulation_gpu(struct Point* points, int num_points, struct Triangle* triangles, int* num_triangles) {
	struct Point* d_points;
	struct Triangle* d_triangles;
	int* d_num_triangles;

	int size_points = sizeof(struct Point) * num_points;
	int size_triangles = sizeof(struct Triangle) * max_num_triangles;

	hipMalloc((void**) &d_points, size_points);
	hipMalloc((void**) &d_triangles, size_triangles);
	hipMalloc((void**) &d_num_triangles, sizeof(int));

	hipMemcpy(d_points, points, size_points, hipMemcpyHostToDevice);
	hipMemcpy(d_num_triangles, num_triangles, sizeof(int), hipMemcpyHostToDevice);

	int collapsed_points = num_points * (num_points - 1) * (num_points - 2);
	int dimGrid = (collapsed_points + (TPB-1)) / TPB; // amount of blocks of size TPB
	int dimBlock = TPB; // int multiple of 32 (warp size) (1024 maximum) try values 128-512

    delaunay_triangulation<<<dimGrid, dimBlock>>>(d_points, num_points, d_triangles, d_num_triangles);

	hipDeviceSynchronize();

	hipMemcpy(num_triangles, d_num_triangles, sizeof(int), hipMemcpyDeviceToHost);
	size_triangles = sizeof(struct Triangle) * *num_triangles;
	hipMemcpy(triangles, d_triangles, size_triangles, hipMemcpyDeviceToHost);
	
	hipFree(d_points); hipFree(d_triangles); hipFree(d_num_triangles);
}

__global__ void save_triangulation_image(struct Point* points, int num_points, struct Triangle* triangles, int num_triangles, int width, int height, double* image) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	int i = idx / width;
	int j = idx % width;

	struct Point pixel;
	double alpha, beta, gamma;

	pixel.x = i;
	pixel.y = j;
	image[pixel(i, j, width)] = -1.0;

	for(int k=0; k<num_triangles; k++) {
		barycentric_coordinates(&triangles[k], &pixel, &alpha, &beta, &gamma);
		if (alpha > 0 && beta > 0 && gamma > 0) {
			image[pixel(i, j, width)] = alpha*(triangles[k].p1.value) + beta*(triangles[k].p2.value) + gamma*(triangles[k].p3.value);
			break;
		}
	}
	
	for(int k=0; k<num_points; k++) {
		if(inside_square(&points[k], &pixel)) {
			image[pixel(i, j, width)] = 101.f;
			break;
		}
	}
}

/*Wraper function to launch the CUDA kernel to compute delaunay triangulation. 
Remember to store an image of int's between 0 and 100, where points store 101, and empty areas -1, and points inside triangle the average of value */
void save_triangulation_image_gpu(struct Point* points, int num_points, struct Triangle* triangles, int num_triangles, int width, int height) {
    //create structures
	int pixels = width * height;
    double* image = (double *) malloc(sizeof(double)*pixels);
	double *d_image;
	struct Point* d_points;
	struct Triangle* d_triangles;

	int size_points = sizeof(struct Point) * num_points;
	int size_triangles = sizeof(struct Triangle) * max_num_triangles;

	hipMalloc((void**) &d_points, size_points);
	hipMalloc((void**) &d_triangles, size_triangles);
	hipMalloc((void**) &d_image, sizeof(double)*pixels);

	int dimGrid = (pixels + (TPB-1)) / TPB; // amount of blocks of size TPB
	int dimBlock = TPB; // int multiple of 32 (warp size) (1024 maximum) try values 128-512
	
	save_triangulation_image<<<dimGrid, dimBlock>>>(d_points, num_points, d_triangles, num_triangles, width, height, d_image);

	hipDeviceSynchronize();

	hipMemcpy(image, d_image, pixels * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_points); hipFree(d_triangles); hipFree(d_image);
	
    //write image
    save_image("image.txt", width, height, image);

    //free structures
    free(image);
}

void printCudaInfo() {
    int devNo = 0;
    printf("\n------------------------------------------------------------------\n");
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);
    printf("Maximum grid size is: (");
    for (int i = 0; i < 3; i++)
        printf("%d, ", iProp.maxGridSize[i]);
    printf(")\n");
    printf("Maximum block dim is: (");
    for (int i = 0; i < 3; i++)
        printf("%d, ", iProp.maxThreadsDim[i]);
    printf(")\n");
    printf("Max threads per block: %d\n", iProp.maxThreadsPerBlock);
    printf("------------------------------------------------------------------\n\n");
}

extern "C" int delaunay(int num_points, int width, int height) {
    printCudaInfo();
    
    float time = 0.f;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

    max_num_triangles = num_points*30;
    struct Point * points = (struct Point *) malloc(sizeof(struct Point)*num_points);
    struct Triangle * triangles = (struct Triangle *) malloc(sizeof(struct Triangle)*max_num_triangles);
    printf("Maximum allowed number of triangles = %d\n", max_num_triangles);
    
    init_points(points, num_points, width, height);

    hipEventRecord(start);
    count_close_points_gpu(points, num_points);
	hipEventRecord(end);
	hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("Counting close points: %f\n", time/1000.f);

    int num_triangles = 0;
    hipEventRecord(start);
    delaunay_triangulation_gpu(points, num_points, triangles, &num_triangles);
    hipEventRecord(end);
	hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("Delaunay triangulation: %f\n", time/1000.f);

    printf("Number of generated triangles = %d\n", num_triangles);

    hipEventRecord(start);
    save_triangulation_image_gpu(points, num_points, triangles, num_triangles, width, height);
	hipEventRecord(end);
	hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("Generate image: %f\n", time/1000.f);

    //Free memory
    free(points);
    free(triangles);

    return 0;
}
    